#include <iostream>
#include <algorithm>
#include <hip/hip_runtime.h>


using std::swap;
using std::cout;

#define BLOCK_SIZE 128
#define CHECK(call)                                                            \
  {                                                                            \
    const hipError_t error = call;                                            \
    if (error != hipSuccess) {                                                \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                   \
      fprintf(stderr, "code: %d, reason: %s\n", error,                         \
              hipGetErrorString(error));                                      \
      exit(1);                                                                 \
    }                                                                          \
  }

struct Node {
  int num_nodes;          // number of nodes in the subtree
  Node *left;             // left subtree
  Node *right;            // right subtree
  float total_length;     // total length of the subtree
  float branch_length[2]; // lengths of left and right subtrees
  Node(int _num_nodes, float _length, Node *_left, Node *_right, float length1,
       float length2)
      : num_nodes(_num_nodes), left(_left), right(_right),
        total_length(_length) {
    branch_length[0] = length1;
    branch_length[1] = length2;
  }
};

__global__ void getMin(float *input, int *input_idx, int n, float *output_val,
                       int *output_idx) {
  __shared__ float smem_val[BLOCK_SIZE];
  __shared__ int smem_idx[BLOCK_SIZE];

  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int i = tx + bx * BLOCK_SIZE * 8;

  float min_val = INFINITY;
  int min_idx = i;

  if (i < n) {
    float a1, a2, a3, a4, a5, a6, a7, a8;
    a1 = input[i];

    a2 = (i + BLOCK_SIZE) < n ? input[i + BLOCK_SIZE] : INFINITY;

    a3 = (i + 2 * BLOCK_SIZE) < n ? input[i + 2 * BLOCK_SIZE] : INFINITY;

    a4 = (i + 3 * BLOCK_SIZE) < n ? input[i + 3 * BLOCK_SIZE] : INFINITY;

    a5 = (i + 4 * BLOCK_SIZE) < n ? input[i + 4 * BLOCK_SIZE] : INFINITY;

    a6 = (i + 5 * BLOCK_SIZE) < n ? input[i + 5 * BLOCK_SIZE] : INFINITY;

    a7 = (i + 6 * BLOCK_SIZE) < n ? input[i + 6 * BLOCK_SIZE] : INFINITY;

    a8 = (i + 7 * BLOCK_SIZE) < n ? input[i + 7 * BLOCK_SIZE] : INFINITY;

    min_val = a1;
    min_idx = i;
    if (a2 < min_val) {
      min_val = a2;
      min_idx = i + BLOCK_SIZE;
    }
    if (a3 < min_val) {
      min_val = a3;
      min_idx = i + 2 * BLOCK_SIZE;
    }
    if (a4 < min_val) {
      min_val = a4;
      min_idx = i + 3 * BLOCK_SIZE;
    }
    if (a5 < min_val) {
      min_val = a5;
      min_idx = i + 4 * BLOCK_SIZE;
    }
    if (a6 < min_val) {
      min_val = a6;
      min_idx = i + 5 * BLOCK_SIZE;
    }
    if (a7 < min_val) {
      min_val = a7;
      min_idx = i + 6 * BLOCK_SIZE;
    }
    if (a8 < min_val) {
      min_val = a8;
      min_idx = i + 7 * BLOCK_SIZE;
    }
  }

  smem_val[tx] = min_val;
  smem_idx[tx] = min_idx;
  __syncthreads();

  // in-place reduction in shared memory
  if (blockDim.x >= 1024 && tx < 512 && smem_val[tx + 512] < smem_val[tx]) {
    smem_val[tx] = min_val = smem_val[tx + 512];
    smem_idx[tx] = min_idx = smem_idx[tx + 512];
  }
  __syncthreads();

  if (blockDim.x >= 512 && tx < 256 && smem_val[tx + 256] < smem_val[tx]) {
    smem_val[tx] = min_val = smem_val[tx + 256];
    smem_idx[tx] = min_idx = smem_idx[tx + 256];
  }
  __syncthreads();

  if (blockDim.x >= 256 && tx < 128 && smem_val[tx + 128] < smem_val[tx]) {
    smem_val[tx] = min_val = smem_val[tx + 128];
    smem_idx[tx] = min_idx = smem_idx[tx + 128];
  }
  __syncthreads();

  if (blockDim.x >= 128 && tx < 64 && smem_val[tx + 64] < smem_val[tx]) {
    smem_val[tx] = min_val = smem_val[tx + 64];
    smem_idx[tx] = min_idx = smem_idx[tx + 64];
  }
  __syncthreads();

  // unrolling warp
  if (tx < 32) {
    volatile float *vsmem_val = smem_val;
    volatile int *vsmem_idx = smem_idx;
    if (vsmem_val[tx + 32] < vsmem_val[tx]) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 32];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 32];
    }
    if (vsmem_val[tx + 16] < vsmem_val[tx]) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 16];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 16];
    }
    if (vsmem_val[tx + 8] < vsmem_val[tx]) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 8];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 8];
    }
    if (vsmem_val[tx + 4] < vsmem_val[tx]) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 4];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 4];
    }
    if (vsmem_val[tx + 2] < vsmem_val[tx]) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 2];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 2];
    }
    if (vsmem_val[tx + 1] < vsmem_val[tx]) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 1];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 1];
    }
  }

  if (tx == 0) {
    output_val[bx] = min_val;
    output_idx[bx] = (input_idx == nullptr) ? min_idx : input_idx[min_idx];
  }
}

__global__ void update(float *mat, int n, int idx1, int idx2, int num_nodes1,
                       int num_nodes2) {
  int tx = threadIdx.x;
  int i = tx + blockDim.x * blockIdx.x;
  if (i >= n) {
    return;
  }
  float val = mat[n * idx1 + i];
  if (isinf(val)) {
    return;
  }
  int total_nodes = num_nodes1 + num_nodes2;
  float new_val =
      (val * num_nodes1 + mat[n * idx2 + i] * num_nodes2) / total_nodes;
  mat[n * idx1 + i] = new_val;
  mat[n * idx2 + i] = INFINITY;
  mat[n * i + idx1] = new_val;
  mat[n * i + idx2] = INFINITY;
}

class UPGMA {
public:
  UPGMA(float *_mat, int _num_seqs) {
    h_mat = _mat;
    num_seqs = _num_seqs;

    int n = num_seqs * num_seqs;
    int n_out_level0 = ceil((float)n / (BLOCK_SIZE * 8));
    int n_out_level1 = ceil((float)n_out_level0 / (BLOCK_SIZE * 8));

    // Allocate host variables
    h_val_level1 = (float *)malloc(sizeof(float) * n_out_level1);
    h_idx_level1 = (int *)malloc(sizeof(int) * n_out_level1);

    // Allocate device variables
    CHECK(hipMalloc((void **)&d_mat, sizeof(float) * n));
    CHECK(hipMalloc((void **)&d_val_level0, sizeof(float) * n_out_level0));
    CHECK(hipMalloc((void **)&d_idx_level0, sizeof(int) * n_out_level0));
    CHECK(hipMalloc((void **)&d_val_level1, sizeof(float) * n_out_level1));
    CHECK(hipMalloc((void **)&d_idx_level1, sizeof(int) * n_out_level1));

    // Copy from host to device
    CHECK(hipMemcpy(d_mat, h_mat, sizeof(float) * n, hipMemcpyHostToDevice));

    nodes = new Node *[num_seqs];
    for (int i = 0; i < num_seqs; ++i) {
      nodes[i] = new Node(1, 0.0f, nullptr, nullptr, 0.0f, 0.0f);
    }

    for (int remain = num_seqs; remain >= 2; --remain) {
      // Reduction round 1
      getMin<<<n_out_level0, BLOCK_SIZE>>>(d_mat, nullptr, n, d_val_level0,
                                           d_idx_level0);

      CHECK(hipDeviceSynchronize());

      // Reduction round 2
      getMin<<<n_out_level1, BLOCK_SIZE>>>(
          d_val_level0, d_idx_level0, n_out_level0, d_val_level1, d_idx_level1);

      CHECK(hipDeviceSynchronize());

      // Copy results and indexes back
      CHECK(hipMemcpy(h_val_level1, d_val_level1, sizeof(float) * n_out_level1,
                       hipMemcpyDeviceToHost));
      CHECK(hipMemcpy(h_idx_level1, d_idx_level1, sizeof(int) * n_out_level1,
                       hipMemcpyDeviceToHost));

      float val = h_val_level1[0];
      int idx = h_idx_level1[0];
      for (int i = 0; i < n_out_level1; ++i) {
        if (h_val_level1[i] < val) {
          val = h_val_level1[i];
          idx = h_idx_level1[i];
        }
      }

      int idx1 = idx / num_seqs;
      int idx2 = idx % num_seqs;
      if (idx1 > idx2) {
        swap(idx1, idx2);
      }
     
      update<<<num_seqs, BLOCK_SIZE>>>(d_mat, num_seqs, idx1, idx2,
                                       nodes[idx1]->num_nodes,
                                       nodes[idx2]->num_nodes);

      float length = val;
      root = new Node(nodes[idx1]->num_nodes + nodes[idx2]->num_nodes,
                      length / 2, nodes[idx1], nodes[idx2],
                      length / 2 - nodes[idx1]->total_length,
                      length / 2 - nodes[idx2]->total_length);

      nodes[idx1] = root;
      nodes[idx2] = nullptr;

      CHECK(hipDeviceSynchronize());
    }
  }

  ~UPGMA() {
    delete[] nodes;
    // Free device
    CHECK(hipFree(d_mat));
    CHECK(hipFree(d_val_level0));
    CHECK(hipFree(d_idx_level0));
    CHECK(hipFree(d_val_level1));
    CHECK(hipFree(d_idx_level1));

    // Free host
    free(h_val_level1);
    free(h_idx_level1);
    cleanup(root);
  }

  void print() {
    print(root);
    cout << "\n";
  }

private:
  float *h_mat;
  float *h_val_level1; // Store result values after level 1 reduction for final
                       // reduction
  int *h_idx_level1;   // Store result indexes after level 1 reduction for final
                       // resuction

  float *d_mat;                       // Device matrix
  float *d_val_level0, *d_val_level1; // Device result values
  int *d_idx_level0, *d_idx_level1;   // Device index values

  int num_seqs;
  Node **nodes;
  Node *root;

  void cleanup(Node *node) {
    if (node == nullptr) {
      return;
    }
    cleanup(node->left);
    cleanup(node->right);
    delete node;
  }

  void print(Node *node) {
    // Reach the leaf
    if (node->left == nullptr && node->right == nullptr) {
      return;
    }
    cout << "(";
    print(node->left);
    cout << ": " << node->branch_length[0] << ", ";
    print(node->right);
    cout << ": " << node->branch_length[1] << ")";
  }
};

int main() {
  /*
  const int num_seqs = 7;
  float h_a[num_seqs][num_seqs]{
      {INFINITY, 19.0f, 27.0f, 8.0f, 33.0f, 18.0f, 13.0f},
      {19.0f, INFINITY, 31.0f, 18.0f, 36.0f, 1.0f, 13.0f},
      {27.0f, 31.0f, INFINITY, 26.0f, 41.0f, 32.0f, 29.0f},
      {8.0f, 18.0f, 26.0f, INFINITY, 31.0f, 17.0f, 14.0f},
      {33.0f, 36.0f, 41.0f, 31.0f, INFINITY, 35.0f, 28.0f},
      {18.0f, 1.0f, 32.0f, 17.0f, 35.0f, INFINITY, 12.0f},
      {13.0f, 13.0f, 29.0f, 14.0f, 28.0f, 12.0f, INFINITY}}; */

  const int num_seqs = 6;
  float h_a[num_seqs][num_seqs]{
      {INFINITY, 2, 4, 6, 6, 8}, 
      {2, INFINITY, 4, 6, 6, 8},
      {4, 4, INFINITY, 6, 6, 8}, 
      {6, 6, 6, INFINITY, 4, 8},
      {6, 6, 6, 4, INFINITY, 8}, 
      {8, 8, 8, 8, 8, INFINITY}};

  UPGMA upgma((float *)h_a, num_seqs);
  upgma.print();
  return 0;
}
