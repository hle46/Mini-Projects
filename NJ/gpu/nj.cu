#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>
#include <vector>
#include <cassert>
#include <cmath>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using std::swap;
using std::cout;
using std::vector;

#define BLOCK_SIZE 128
#define Q_BLOCK_SIZE 16

#define CHECK(call)                                                            \
  {                                                                            \
    const hipError_t error = call;                                            \
    if (error != hipSuccess) {                                                \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                   \
      fprintf(stderr, "code: %d, reason: %s\n", error,                         \
              hipGetErrorString(error));                                      \
      exit(1);                                                                 \
    }                                                                          \
  }

struct Node {
  Node() = default;
  Node(Node *left, Node *right, float length1, float length2)
      : childs{left, right}, branch_length{length1, length2} {}
  ~Node() = default;
  vector<Node *> childs;
  vector<float> branch_length;
};

__global__ void sum_level0(float *input, int n_e, int n_b, float *output_val) {
  __shared__ float smem_val[BLOCK_SIZE];

  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int i = (bx / n_b) * n_e + tx +
          (bx % n_b) * BLOCK_SIZE * 8; // (bx / n_b) * n_e is offset
  int n = ((bx / n_b) + 1) * n_e;
  float val = 0.0f;

  if (i < n) {
    float a1, a2, a3, a4, a5, a6, a7, a8;
    a1 = input[i];
    a1 = isinf(a1) ? 0.0f : a1;

    a2 = (i + BLOCK_SIZE) < n ? input[i + BLOCK_SIZE] : 0.0f;
    a2 = isinf(a2) ? 0.0f : a2;

    a3 = (i + 2 * BLOCK_SIZE) < n ? input[i + 2 * BLOCK_SIZE] : 0.0f;
    a3 = isinf(a3) ? 0.0f : a3;

    a4 = (i + 3 * BLOCK_SIZE) < n ? input[i + 3 * BLOCK_SIZE] : 0.0f;
    a4 = isinf(a4) ? 0.0f : a4;

    a5 = (i + 4 * BLOCK_SIZE) < n ? input[i + 4 * BLOCK_SIZE] : 0.0f;
    a5 = isinf(a5) ? 0.0f : a5;

    a6 = (i + 5 * BLOCK_SIZE) < n ? input[i + 5 * BLOCK_SIZE] : 0.0f;
    a6 = isinf(a6) ? 0.0f : a6;

    a7 = (i + 6 * BLOCK_SIZE) < n ? input[i + 6 * BLOCK_SIZE] : 0.0f;
    a7 = isinf(a7) ? 0.0f : a7;

    a8 = (i + 7 * BLOCK_SIZE) < n ? input[i + 7 * BLOCK_SIZE] : 0.0f;
    a8 = isinf(a8) ? 0.0f : a8;

    val = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;
  }

  smem_val[tx] = val;
  __syncthreads();

  // in-place reduction in shared memory
  if (blockDim.x >= 1024 && tx < 512) {
    smem_val[tx] = val = val + smem_val[tx + 512];
  }
  __syncthreads();

  if (blockDim.x >= 512 && tx < 256) {
    smem_val[tx] = val = val + smem_val[tx + 256];
  }
  __syncthreads();

  if (blockDim.x >= 256 && tx < 128) {
    smem_val[tx] = val = val + smem_val[tx + 128];
  }
  __syncthreads();

  if (blockDim.x >= 128 && tx < 64) {
    smem_val[tx] = val = val + smem_val[tx + 64];
  }
  __syncthreads();

  // unrolling warp
  if (tx < 32) {
    volatile float *vsmem_val = smem_val;
    vsmem_val[tx] = val = val + vsmem_val[tx + 32];
    vsmem_val[tx] = val = val + vsmem_val[tx + 16];
    vsmem_val[tx] = val = val + vsmem_val[tx + 8];
    vsmem_val[tx] = val = val + vsmem_val[tx + 4];
    vsmem_val[tx] = val = val + vsmem_val[tx + 2];
    vsmem_val[tx] = val = val + vsmem_val[tx + 1];
  }

  if (tx == 0) {
    output_val[(bx / n_b) + (bx % n_b) * n_e] = val;
  }
}

__global__ void sum_level1(float *input, int n_e, int n_b, float *output) {
  int tx = threadIdx.x;
  int i = tx + blockIdx.x * blockDim.x;
  float val = 0.0f;
  if (i >= n_e) {
    return;
  }
  for (int j = 0; j < n_b; ++j) {
    val += input[i + j * n_e];
  }
  printf("i: %d, val: %f\n", i, val);
  output[i] = val;
}

__global__ void calculate_q(float *mat, float *s, int n, int remain, float *q) {
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int i = ty + blockIdx.y * blockDim.y;
  int j = tx + blockIdx.x * blockDim.x;
  if (i >= n || j >= n) {
    return;
  }
  __shared__ float smem_i[Q_BLOCK_SIZE];
  __shared__ float smem_j[Q_BLOCK_SIZE];
  if (tx == 0) {
    smem_i[ty] = s[i];
  }
  if (ty == 0) {
    smem_j[tx] = s[j];
  }
  __syncthreads();

  float val = mat[i * n + j];
  q[i * n + j] =
      isinf(val) ? INFINITY : ((remain - 2) * val - smem_i[ty] - smem_j[tx]);
  printf("i: %d, j: %d, q: %f\n", i, j, q[i * n + j]);
}

__global__ void update(float *mat, int n, int idx1, int idx2, int num_nodes1,
                       int num_nodes2) {
  int tx = threadIdx.x;
  int i = tx + blockDim.x * blockIdx.x;
  if (i >= n) {
    return;
  }
  float val = mat[n * idx1 + i];
  if (isinf(val)) {
    return;
  }
  int total_nodes = num_nodes1 + num_nodes2;
  float new_val =
      (val * num_nodes1 + mat[n * idx2 + i] * num_nodes2) / total_nodes;
  mat[n * idx1 + i] = new_val;
  mat[n * idx2 + i] = INFINITY;
  mat[n * i + idx1] = new_val;
  mat[n * i + idx2] = INFINITY;
}

class NJ {
public:
  NJ(float *_mat, int _num_seqs) : h_mat{_mat}, num_seqs{_num_seqs} {
    vector<Node *> nodes(num_seqs);
    for (int i = 0; i < num_seqs; ++i) {
      nodes[i] = new Node(nullptr, nullptr, 0.0f, 0.0f);
    }

    int n = num_seqs * num_seqs;
    // number of blocks to calculate a row
    int n_blocks_per_row = ceil(num_seqs / (float)(BLOCK_SIZE * 8));
    int n_out = n_blocks_per_row * num_seqs;

    // Allocate device variables
    float *d_mat;                   // Device matrix
    float *d_q;                     // Device q matrix
    float *d_s_level0, *d_s_level1; // Device s matrix
    CHECK(hipMalloc((void **)&d_mat, sizeof(float) * n));
    CHECK(hipMalloc((void **)&d_q, sizeof(float) * n));
    CHECK(hipMalloc((void **)&d_s_level0, sizeof(float) * n_out));
    CHECK(hipMalloc((void **)&d_s_level1, sizeof(float) * num_seqs));

    CHECK(hipMemcpy(d_mat, h_mat, sizeof(float) * n, hipMemcpyHostToDevice));

    float *q = (float *)malloc(sizeof(float) * num_seqs * num_seqs);
    float *s = (float *)malloc(sizeof(float) * num_seqs);
    int root_idx = -1;
    for (int remain = num_seqs; remain > 2; --remain) {
      // Calculate sums over row on GPU
      sum_level0<<<n_out, BLOCK_SIZE>>>(d_mat, num_seqs, n_blocks_per_row,
                                        d_s_level0);
      CHECK(hipDeviceSynchronize());

      sum_level1<<<ceil(num_seqs / (float)BLOCK_SIZE), BLOCK_SIZE>>>(
          d_s_level0, num_seqs, n_blocks_per_row, d_s_level1);
      CHECK(hipDeviceSynchronize());

      // calculate sums over row
      for (int i = 0; i < num_seqs; ++i) {
        s[i] = 0.0f;
        for (int j = 0; j < num_seqs; ++j) {
          s[i] +=
              isinf(h_mat[i * num_seqs + j]) ? 0.0f : h_mat[i * num_seqs + j];
        }
      }

      // Calculate q matrix on GPU
      calculate_q<<<dim3(ceil(num_seqs / (float)Q_BLOCK_SIZE),
                         ceil(num_seqs / (float)Q_BLOCK_SIZE), 1),
                    dim3(Q_BLOCK_SIZE, Q_BLOCK_SIZE, 1)>>>(d_mat, d_s_level1,
                                                       num_seqs, remain, d_q);

      CHECK(hipDeviceSynchronize());

      // calculate q matrix;
      for (int i = 0; i < num_seqs; ++i) {
        for (int j = 0; j < num_seqs; ++j) {
          q[i * num_seqs + j] =
              isinf(h_mat[i * num_seqs + j])
                  ? INFINITY
                  : (remain - 2) * h_mat[i * num_seqs + j] - s[i] - s[j];
        }
      }

      for (int i = 0; i < num_seqs; ++i) {
        for (int j = 0; j < num_seqs; ++j) {
          cout << q[i * num_seqs + j] << ",\t";
        }
        cout << "\n";
      }
      cout << "--------------------------------------\n";

      int idx = getMinIdx(q, num_seqs * num_seqs);
      int idx1 = idx / num_seqs;
      int idx2 = idx % num_seqs;
      if (idx1 > idx2) {
        swap(idx1, idx2);
      }
      cout << idx1 << ", " << idx2 << "\n";
      float length = h_mat[idx1 * num_seqs + idx2];

      float branch_length1 =
          length / 2 + (s[idx1] - s[idx2]) / ((remain - 2) * 2);
      float branch_length2 = length - branch_length1;
      root = new Node(nodes[idx1], nodes[idx2], branch_length1, branch_length2);
      update(idx1, idx2);

      for (int i = 0; i < num_seqs; ++i) {
        for (int j = 0; j < num_seqs; ++j) {
          cout << h_mat[num_seqs * i + j] << ",\t";
        }
        cout << "\n";
      }
      cout << "--------------------------------------\n";
      root_idx = idx1;
      nodes[idx1] = root;
      nodes[idx2] = nullptr;
    }

    Node *other_root = nullptr;
    int other_root_idx = -1;
    for (int i = 0; i < num_seqs; ++i) {
      if (nodes[i] != nullptr && nodes[i] != root) {
        other_root = nodes[i];
        other_root_idx = i;
        break;
      }
    }
    root->childs.push_back(other_root);
    root->branch_length.push_back(h_mat[root_idx * num_seqs + other_root_idx]);

    // Free memory
    free(q);
  }

  void print() {
    print(root);
    cout << "\n";
  }

private:
  float *h_mat;
  int num_seqs;
  Node *root = nullptr;

  int getMinIdx(float *a, int n) {
    float val = INFINITY;
    int idx = -1;
    for (int i = 0; i < n; ++i) {
      if (a[i] < val) {
        idx = i;
        val = a[i];
      }
    }
    return idx;
  }

  void update(int idx1, int idx2) {
    float d = h_mat[num_seqs * idx1 + idx2];
    for (int i = 0; i < num_seqs; ++i) {
      float val = h_mat[num_seqs * idx1 + i];
      if (isinf(val)) {
        continue;
      }
      float new_val = (val + h_mat[num_seqs * idx2 + i] - d) / 2;
      h_mat[num_seqs * idx1 + i] = new_val;
      h_mat[num_seqs * idx2 + i] = INFINITY;
      h_mat[num_seqs * i + idx1] = new_val;
      h_mat[num_seqs * i + idx2] = INFINITY;
    }
  }

  void cleanup(Node *node) {
    if (node == nullptr) {
      return;
    }
    int num_childs = node->childs.size();
    for (int i = 0; i < num_childs; ++i) {
      cleanup(node->childs[i]);
    }
    delete node;
  }

  void print(Node *node) {
    int num_childs = node->childs.size();
    // Reach the leaf
    if (num_childs == 2 && node->childs[0] == nullptr &&
        node->childs[1] == nullptr) {
      return;
    }
    cout << "(";
    for (int i = 0; i < num_childs - 1; ++i) {
      print(node->childs[i]);
      cout << ": " << node->branch_length[i] << ", ";
    }
    print(node->childs[num_childs - 1]);
    cout << ": " << node->branch_length[num_childs - 1] << ")";
  }
};

int main() {
  const int num_seqs = 5;
  float a[num_seqs][num_seqs]{{INFINITY, 5.0f, 9.0f, 9.0f, 8.0f},
                              {5.0f, INFINITY, 10.0f, 10.0f, 9.0f},
                              {9.0f, 10.0f, INFINITY, 8.0f, 7.0f},
                              {9.0f, 10.0f, 8.0f, INFINITY, 3.0f},
                              {8.0f, 9.0f, 7.0f, 3.0f, INFINITY}};

  assert(num_seqs > 2);
  NJ nj((float *)a, num_seqs);
  nj.print();
  return 0;
}
