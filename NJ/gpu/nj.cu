#include <iostream>
#include <limits>
#include <vector>
#include <cassert>
#include <cmath>
#include <algorithm>
#include <hip/hip_runtime.h>


using std::swap;
using std::cout;
using std::vector;

#define BLOCK_SIZE 128 // Block size should be multiple of 64
#define Q_BLOCK_SIZE 32

#define CHECK(call)                                                            \
  {                                                                            \
    const hipError_t error = call;                                            \
    if (error != hipSuccess) {                                                \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                   \
      fprintf(stderr, "code: %d, reason: %s\n", error,                         \
              hipGetErrorString(error));                                      \
      exit(1);                                                                 \
    }                                                                          \
  }

struct Node {
  Node() = default;
  Node(int _id, Node *left, Node *right, float length1, float length2)
      : id{_id}, childs{left, right}, branch_length{length1, length2} {}
  ~Node() = default;
  int id;
  vector<Node *> childs;
  vector<float> branch_length;
};

template <unsigned int blockSize>
__global__ void sum_level0(float *input, int n_e, int n_b, float *output_val) {
  __shared__ float smem_val[blockSize];

  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int i = (bx / n_b) * n_e + tx +
          (bx % n_b) * blockSize * 8; // (bx / n_b) * n_e is offset
  int n = ((bx / n_b) + 1) * n_e;
  float val = 0.0f;

  if (i < n) {
    float a1, a2, a3, a4, a5, a6, a7, a8;
    a1 = input[i];
    a1 = isinf(a1) ? 0.0f : a1;

    a2 = (i + blockSize) < n ? input[i + blockSize] : 0.0f;
    a2 = isinf(a2) ? 0.0f : a2;

    a3 = (i + 2 * blockSize) < n ? input[i + 2 * blockSize] : 0.0f;
    a3 = isinf(a3) ? 0.0f : a3;

    a4 = (i + 3 * blockSize) < n ? input[i + 3 * blockSize] : 0.0f;
    a4 = isinf(a4) ? 0.0f : a4;

    a5 = (i + 4 * blockSize) < n ? input[i + 4 * blockSize] : 0.0f;
    a5 = isinf(a5) ? 0.0f : a5;

    a6 = (i + 5 * blockSize) < n ? input[i + 5 * blockSize] : 0.0f;
    a6 = isinf(a6) ? 0.0f : a6;

    a7 = (i + 6 * blockSize) < n ? input[i + 6 * blockSize] : 0.0f;
    a7 = isinf(a7) ? 0.0f : a7;

    a8 = (i + 7 * blockSize) < n ? input[i + 7 * blockSize] : 0.0f;
    a8 = isinf(a8) ? 0.0f : a8;

    val = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;
  }

  smem_val[tx] = val;
  __syncthreads();

  // in-place reduction in shared memory
  if (blockSize >= 1024 && tx < 512) {
    smem_val[tx] = val = val + smem_val[tx + 512];
  }
  __syncthreads();

  if (blockSize >= 512 && tx < 256) {
    smem_val[tx] = val = val + smem_val[tx + 256];
  }
  __syncthreads();

  if (blockSize >= 256 && tx < 128) {
    smem_val[tx] = val = val + smem_val[tx + 128];
  }
  __syncthreads();

  if (blockSize >= 128 && tx < 64) {
    smem_val[tx] = val = val + smem_val[tx + 64];
  }
  __syncthreads();

  // unrolling warp
  if (tx < 32) {
    volatile float *vsmem_val = smem_val;
    if (blockSize >= 64) {
      vsmem_val[tx] = val = val + vsmem_val[tx + 32];
    }
    if (blockSize >= 32) {
      vsmem_val[tx] = val = val + vsmem_val[tx + 16];
    }
    if (blockSize >= 16) {
      vsmem_val[tx] = val = val + vsmem_val[tx + 8];
    }
    if (blockSize >= 8) {
      vsmem_val[tx] = val = val + vsmem_val[tx + 4];
    }
    if (blockSize >= 4) {
      vsmem_val[tx] = val = val + vsmem_val[tx + 2];
    }
    if (blockSize >= 2) {
      vsmem_val[tx] = val = val + vsmem_val[tx + 1];
    }
  }

  if (tx == 0) {
    // printf("Block: %d, val: %f\n", bx, val);
    output_val[(bx / n_b) + (bx % n_b) * n_e] = val;
  }
}

__global__ void sum_level1(float *input, int n_e, int n_b, float *output) {
  int tx = threadIdx.x;
  int i = tx + blockIdx.x * blockDim.x;
  float val = 0.0f;
  if (i >= n_e) {
    return;
  }
  for (int j = 0; j < n_b; ++j) {
    val += input[i + j * n_e];
  }
  output[i] = val;
  // printf("%d, %f\n", i, val);
}

__global__ void calculate_q(float *mat, float *s, int n, int remain, float *q) {
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int i = ty + blockIdx.y * blockDim.y;
  int j = tx + blockIdx.x * blockDim.x;
  if (i >= n || j >= n) {
    return;
  }
  /*
  __shared__ float smem_i[Q_BLOCK_SIZE];
  __shared__ float smem_j[Q_BLOCK_SIZE];
  if (tx == 0) {
    smem_i[ty] = s[i];
  }
  if (ty == 0) {
    smem_j[tx] = s[j];
  }
  __syncthreads();*/

  float val = mat[i * n + j];
  q[i * n + j] = isinf(val) ? INFINITY : ((remain - 2) * val - s[i] - s[j]);
}

template <unsigned int blockSize>
__global__ void getMin1(float *input, int *input_idx, int n, float *s, int m,
                        int remain, float *output_val, int *output_idx) {
  __shared__ float smem_val[blockSize];
  __shared__ int smem_idx[blockSize];

  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int i = tx + bx * blockSize * 8;

  float min_val = INFINITY;
  int min_idx = i;

  if (i < n) {
    float a1, a2, a3, a4, a5, a6, a7, a8;
    a1 = input[i];
    a1 = isinf(a1) ? INFINITY : ((remain - 2) * a1 - s[i / m] - s[i % m]);
    min_val = a1;
    min_idx = i;

    i += blockSize;
    a2 = i < n ? input[i] : INFINITY;
    a2 = isinf(a2) ? INFINITY : ((remain - 2) * a2 - s[i / m] - s[i % m]);
    if (a2 < min_val) {
      min_val = a2;
      min_idx = i;
    }

    i += blockSize;
    a3 = i < n ? input[i] : INFINITY;
    a3 = isinf(a3) ? INFINITY : ((remain - 2) * a3 - s[i / m] - s[i % m]);
    if (a3 < min_val) {
      min_val = a3;
      min_idx = i;
    }

    i += blockSize;
    a4 = i < n ? input[i] : INFINITY;
    a4 = isinf(a4) ? INFINITY : ((remain - 2) * a4 - s[i / m] - s[i % m]);
    if (a4 < min_val) {
      min_val = a4;
      min_idx = i;
    }

    i += blockSize;
    a5 = i < n ? input[i] : INFINITY;
    a5 = isinf(a5) ? INFINITY : ((remain - 2) * a5 - s[i / m] - s[i % m]);
    if (a5 < min_val) {
      min_val = a5;
      min_idx = i;
    }

    i += blockSize;
    a6 = i < n ? input[i] : INFINITY;
    a6 = isinf(a6) ? INFINITY : ((remain - 2) * a6 - s[i / m] - s[i % m]);
    if (a6 < min_val) {
      min_val = a6;
      min_idx = i;
    }

    i += blockSize;
    a7 = i < n ? input[i] : INFINITY;
    a7 = isinf(a7) ? INFINITY : ((remain - 2) * a7 - s[i / m] - s[i % m]);
    if (a7 < min_val) {
      min_val = a7;
      min_idx = i;
    }

    i += blockSize;
    a8 = i < n ? input[i] : INFINITY;
    a8 = isinf(a8) ? INFINITY : ((remain - 2) * a8 - s[i / m] - s[i % m]);
    if (a8 < min_val) {
      min_val = a8;
      min_idx = i;
    }
  }

  smem_val[tx] = min_val;
  smem_idx[tx] = min_idx;
  __syncthreads();

  // in-place reduction in shared memory
  if (blockSize >= 1024 && tx < 512 && smem_val[tx + 512] < min_val) {
    smem_val[tx] = min_val = smem_val[tx + 512];
    smem_idx[tx] = min_idx = smem_idx[tx + 512];
  }
  __syncthreads();

  if (blockSize >= 512 && tx < 256 && smem_val[tx + 256] < min_val) {
    smem_val[tx] = min_val = smem_val[tx + 256];
    smem_idx[tx] = min_idx = smem_idx[tx + 256];
  }
  __syncthreads();

  if (blockSize >= 256 && tx < 128 && smem_val[tx + 128] < min_val) {
    smem_val[tx] = min_val = smem_val[tx + 128];
    smem_idx[tx] = min_idx = smem_idx[tx + 128];
  }
  __syncthreads();

  if (blockSize >= 128 && tx < 64 && smem_val[tx + 64] < min_val) {
    smem_val[tx] = min_val = smem_val[tx + 64];
    smem_idx[tx] = min_idx = smem_idx[tx + 64];
  }
  __syncthreads();

  // unrolling warp
  if (tx < 32) {
    volatile float *vsmem_val = smem_val;
    volatile int *vsmem_idx = smem_idx;
    if (blockSize >= 64 && vsmem_val[tx + 32] < min_val) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 32];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 32];
    }
    if (blockSize >= 32 && vsmem_val[tx + 16] < min_val) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 16];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 16];
    }
    if (blockSize >= 16 && vsmem_val[tx + 8] < min_val) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 8];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 8];
    }
    if (blockSize >= 8 && vsmem_val[tx + 4] < min_val) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 4];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 4];
    }
    if (blockSize >= 4 && vsmem_val[tx + 2] < min_val) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 2];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 2];
    }
    if (blockSize >= 2 && vsmem_val[tx + 1] < min_val) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 1];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 1];
    }
  }

  if (tx == 0) {
    output_val[bx] = min_val;
    output_idx[bx] = (input_idx == nullptr) ? min_idx : input_idx[min_idx];
  }
}

template <unsigned int blockSize>
__global__ void getMin(float *input, int *input_idx, int n, float *output_val,
                       int *output_idx) {
  __shared__ float smem_val[blockSize];
  __shared__ int smem_idx[blockSize];

  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int i = tx + bx * blockSize * 8;

  float min_val = INFINITY;
  int min_idx = i;

  if (i < n) {
    float a1, a2, a3, a4, a5, a6, a7, a8;
    a1 = input[i];

    a2 = (i + blockSize) < n ? input[i + blockSize] : INFINITY;

    a3 = (i + 2 * blockSize) < n ? input[i + 2 * blockSize] : INFINITY;

    a4 = (i + 3 * blockSize) < n ? input[i + 3 * blockSize] : INFINITY;

    a5 = (i + 4 * blockSize) < n ? input[i + 4 * blockSize] : INFINITY;

    a6 = (i + 5 * blockSize) < n ? input[i + 5 * blockSize] : INFINITY;

    a7 = (i + 6 * blockSize) < n ? input[i + 6 * blockSize] : INFINITY;

    a8 = (i + 7 * blockSize) < n ? input[i + 7 * blockSize] : INFINITY;

    min_val = a1;
    min_idx = i;
    if (a2 < min_val) {
      min_val = a2;
      min_idx = i + blockSize;
    }
    if (a3 < min_val) {
      min_val = a3;
      min_idx = i + 2 * blockSize;
    }
    if (a4 < min_val) {
      min_val = a4;
      min_idx = i + 3 * blockSize;
    }
    if (a5 < min_val) {
      min_val = a5;
      min_idx = i + 4 * blockSize;
    }
    if (a6 < min_val) {
      min_val = a6;
      min_idx = i + 5 * blockSize;
    }
    if (a7 < min_val) {
      min_val = a7;
      min_idx = i + 6 * blockSize;
    }
    if (a8 < min_val) {
      min_val = a8;
      min_idx = i + 7 * blockSize;
    }
  }

  smem_val[tx] = min_val;
  smem_idx[tx] = min_idx;
  __syncthreads();

  // in-place reduction in shared memory
  if (blockSize >= 1024 && tx < 512 && smem_val[tx + 512] < min_val) {
    smem_val[tx] = min_val = smem_val[tx + 512];
    smem_idx[tx] = min_idx = smem_idx[tx + 512];
  }
  __syncthreads();

  if (blockSize >= 512 && tx < 256 && smem_val[tx + 256] < min_val) {
    smem_val[tx] = min_val = smem_val[tx + 256];
    smem_idx[tx] = min_idx = smem_idx[tx + 256];
  }
  __syncthreads();

  if (blockSize >= 256 && tx < 128 && smem_val[tx + 128] < min_val) {
    smem_val[tx] = min_val = smem_val[tx + 128];
    smem_idx[tx] = min_idx = smem_idx[tx + 128];
  }
  __syncthreads();

  if (blockSize >= 128 && tx < 64 && smem_val[tx + 64] < min_val) {
    smem_val[tx] = min_val = smem_val[tx + 64];
    smem_idx[tx] = min_idx = smem_idx[tx + 64];
  }
  __syncthreads();

  // unrolling warp
  if (tx < 32) {
    volatile float *vsmem_val = smem_val;
    volatile int *vsmem_idx = smem_idx;
    if (blockSize >= 64 && vsmem_val[tx + 32] < min_val) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 32];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 32];
    }
    if (blockSize >= 32 && vsmem_val[tx + 16] < min_val) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 16];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 16];
    }
    if (blockSize >= 16 && vsmem_val[tx + 8] < min_val) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 8];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 8];
    }
    if (blockSize >= 8 && vsmem_val[tx + 4] < min_val) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 4];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 4];
    }
    if (blockSize >= 4 && vsmem_val[tx + 2] < min_val) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 2];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 2];
    }
    if (blockSize >= 2 && vsmem_val[tx + 1] < min_val) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 1];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 1];
    }
  }

  if (tx == 0) {
    output_val[bx] = min_val;
    output_idx[bx] = (input_idx == nullptr) ? min_idx : input_idx[min_idx];
  }
}

__global__ void update(float *mat, int n, float d, int idx1, int idx2) {
  int tx = threadIdx.x;
  int i = tx + blockDim.x * blockIdx.x;
  if (i >= n) {
    return;
  }
  if (i == idx2) {
    mat[n * idx1 + idx2] = INFINITY;
    mat[n * idx2 + idx1] = INFINITY;
    return;
  }
  float val = mat[n * idx1 + i];
  if (isinf(val)) {
    return;
  }
  float new_val = (val + mat[n * idx2 + i] - d) / 2.0;
  mat[n * idx1 + i] = new_val;
  mat[n * idx2 + i] = INFINITY;
  mat[n * i + idx1] = new_val;
  mat[n * i + idx2] = INFINITY;
}

class NJ {
public:
  NJ(float *_mat, int _num_seqs)
      : h_mat{_mat}, num_seqs{_num_seqs}, root{nullptr} {
    vector<Node *> nodes(num_seqs);
    for (int i = 0; i < num_seqs; ++i) {
      nodes[i] = new Node(i, nullptr, nullptr, 0.0f, 0.0f);
    }

    int n = num_seqs * num_seqs;
    // number of blocks to calculate a row
    int n_blocks_per_row = ceil(num_seqs / (float)(BLOCK_SIZE * 8));
    int n_out = n_blocks_per_row * num_seqs;

    int n_out_level0 = ceil((float)n / (BLOCK_SIZE * 8));
    int n_out_level1 = ceil((float)n_out_level0 / (BLOCK_SIZE * 8));

    // Allocate host variables
    // Result values after level 1 reduction for final reduction
    float *h_val_level1 = (float *)malloc(sizeof(float) * n_out_level1);
    // Result indexes after level 1 reduction for final reduction
    int *h_idx_level1 = (int *)malloc(sizeof(int) * n_out_level1);

    // Allocate device variables
    float *d_mat;                       // Device matrix
    //float *d_q;                         // Device q matrix
    float *d_s_level0, *d_s_level1;     // Device s matrix
    float *d_val_level0, *d_val_level1; // Device min result values
    int *d_idx_level0, *d_idx_level1;   // Device min index values
    CHECK(hipMalloc((void **)&d_mat, sizeof(float) * n));
    //CHECK(cudaMalloc((void **)&d_q, sizeof(float) * n));
    CHECK(hipMalloc((void **)&d_s_level0, sizeof(float) * n_out));
    CHECK(hipMalloc((void **)&d_s_level1, sizeof(float) * num_seqs));
    CHECK(hipMalloc((void **)&d_val_level0, sizeof(float) * n_out_level0));
    CHECK(hipMalloc((void **)&d_idx_level0, sizeof(int) * n_out_level0));
    CHECK(hipMalloc((void **)&d_val_level1, sizeof(float) * n_out_level1));
    CHECK(hipMalloc((void **)&d_idx_level1, sizeof(int) * n_out_level1));

    CHECK(hipMemcpy(d_mat, h_mat, sizeof(float) * n, hipMemcpyHostToDevice));

    //float *q = (float *)malloc(sizeof(float) * num_seqs * num_seqs);
    int root_idx = -1;
    for (int remain = num_seqs; remain > 2; --remain) {
      // Calculate sums over row on GPU
      sum_level0<BLOCK_SIZE><<<n_out, BLOCK_SIZE>>>(
          d_mat, num_seqs, n_blocks_per_row, d_s_level0);
      CHECK(hipDeviceSynchronize());

      sum_level1<<<ceil(num_seqs / (float)BLOCK_SIZE), BLOCK_SIZE>>>(
          d_s_level0, num_seqs, n_blocks_per_row, d_s_level1);
      CHECK(hipDeviceSynchronize());

      // Calculate q matrix on GPU
      /*calculate_q<<<dim3(ceil(num_seqs / (float)Q_BLOCK_SIZE),
                         ceil(num_seqs / (float)Q_BLOCK_SIZE), 1),
                    dim3(Q_BLOCK_SIZE, Q_BLOCK_SIZE, 1)>>>(
		    d_mat, d_s_level1, num_seqs, remain, d_q); */

      //CHECK(cudaDeviceSynchronize());

      /*
      // Copy back device q back to host q to check
      CHECK(cudaMemcpy(q, d_q, sizeof(float) * num_seqs * num_seqs,
      cudaMemcpyDeviceToHost));
      for (int i = 0; i < num_seqs; ++i) {
        for (int j = 0; j < num_seqs; ++j) {
          cout << q[i * num_seqs + j] << ",\t";
        }
        cout << "\n";
      }
      cout << "--------------------------------------\n";*/

      // Get min on GPU
      // Reduction round 1
      getMin1<BLOCK_SIZE><<<n_out_level0, BLOCK_SIZE>>>(
          d_mat, nullptr, n, d_s_level1, num_seqs, remain, d_val_level0,
          d_idx_level0);
      CHECK(hipDeviceSynchronize());

      // Reduction round 2
      getMin<BLOCK_SIZE><<<n_out_level1, BLOCK_SIZE>>>(
          d_val_level0, d_idx_level0, n_out_level0, d_val_level1, d_idx_level1);
      CHECK(hipDeviceSynchronize());

      // Copy results and indexes back
      CHECK(hipMemcpy(h_val_level1, d_val_level1, sizeof(float) * n_out_level1,
                       hipMemcpyDeviceToHost));
      CHECK(hipMemcpy(h_idx_level1, d_idx_level1, sizeof(int) * n_out_level1,
                       hipMemcpyDeviceToHost));

      float val = h_val_level1[0];
      int idx = h_idx_level1[0];
      for (int i = 0; i < n_out_level1; ++i) {
        if (h_val_level1[i] < val) {
          val = h_val_level1[i];
          idx = h_idx_level1[i];
        }
      }

      int idx1 = idx / num_seqs;
      int idx2 = idx % num_seqs;
      if (idx1 > idx2) {
        swap(idx1, idx2);
      }
      // cout << idx1 << ", " << idx2 << "\n";

      float length;
      CHECK(hipMemcpy(&length, &d_mat[idx1 * num_seqs + idx2], sizeof(float),
                       hipMemcpyDeviceToHost));
      float s1, s2;
      CHECK(hipMemcpy(&s1, &d_s_level1[idx1], sizeof(float),
                       hipMemcpyDeviceToHost));
      CHECK(hipMemcpy(&s2, &d_s_level1[idx2], sizeof(float),
                       hipMemcpyDeviceToHost));

      update<<<ceil(num_seqs / (float)BLOCK_SIZE), BLOCK_SIZE>>>(
          d_mat, num_seqs, length, idx1, idx2);

      float branch_length1 = length / 2 + (s1 - s2) / ((remain - 2) * 2);
      float branch_length2 = length - branch_length1;

      root = new Node(-1, nodes[idx1], nodes[idx2], branch_length1,
                      branch_length2);
      root_idx = idx1;
      nodes[idx1] = root;
      nodes[idx2] = nullptr;

      CHECK(hipDeviceSynchronize());

      /*
      // Copy device mat back to host mat to check
      CHECK(cudaMemcpy(h_mat, d_mat, sizeof(float) * num_seqs * num_seqs,
      cudaMemcpyDeviceToHost));
      for (int i = 0; i < num_seqs; ++i) {
        for (int j = 0; j < num_seqs; ++j) {
          cout << h_mat[num_seqs * i + j] << ",\t";
        }
        cout << "\n";
      }
      cout << "--------------------------------------\n"; */
    }

    Node *other_root = nullptr;
    int other_root_idx = -1;
    for (int i = 0; i < num_seqs; ++i) {
      if (nodes[i] != nullptr && nodes[i] != root) {
        other_root = nodes[i];
        other_root_idx = i;
        nodes[i] = nullptr;
        break;
      }
    }

    float length;
    CHECK(hipMemcpy(&length, &d_mat[root_idx * num_seqs + other_root_idx],
                     sizeof(float), hipMemcpyDeviceToHost));

    if (root_idx < other_root_idx) {
      root->childs.push_back(other_root);
      root->branch_length.push_back(length);
    } else {
      other_root->childs.push_back(root);
      other_root->branch_length.push_back(length);
    }

    // Free device memory
    CHECK(hipFree(d_mat));
    //CHECK(cudaFree(d_q));
    CHECK(hipFree(d_s_level0));
    CHECK(hipFree(d_s_level1));
    CHECK(hipFree(d_val_level0));
    CHECK(hipFree(d_idx_level0));
    CHECK(hipFree(d_val_level1));
    CHECK(hipFree(d_idx_level1));

    // Free host memory
    free(h_val_level1);
    free(h_idx_level1);
    //free(q);
  }

  void print() {
    print(root);
    cout << "\n";
  }

private:
  float *h_mat;
  int num_seqs;
  Node *root;

  void cleanup(Node *node) {
    if (node == nullptr) {
      return;
    }
    int num_childs = node->childs.size();
    for (int i = 0; i < num_childs; ++i) {
      cleanup(node->childs[i]);
    }
    delete node;
  }

  void print(Node *node) {
    if (node == nullptr) {
      cout << "Oops! Null pointer\n";
    }
    int num_childs = node->childs.size();
    // Reach the leaf
    if (num_childs == 2 && node->childs[0] == nullptr &&
        node->childs[1] == nullptr) {
      cout << "A" + std::to_string(node->id);
      return;
    }
    cout << "(";
    for (int i = 0; i < num_childs - 1; ++i) {
      print(node->childs[i]);
      cout << ":" << node->branch_length[i] << ",";
    }
    print(node->childs[num_childs - 1]);
    cout << ":" << node->branch_length[num_childs - 1] << ")";
  }
};

int main() {
  /*
  const int num_seqs = 5;
  float a[num_seqs][num_seqs]{{INFINITY, 5.0f, 9.0f, 9.0f, 8.0f},
                              {5.0f, INFINITY, 10.0f, 10.0f, 9.0f},
                              {9.0f, 10.0f, INFINITY, 8.0f, 7.0f},
                              {9.0f, 10.0f, 8.0f, INFINITY, 3.0f},
                              {8.0f, 9.0f, 7.0f, 3.0f, INFINITY}};*/

  const int num_seqs = 2048;
  float *a = new float[num_seqs * num_seqs];
  srand(0);
  for (int i = 0; i < num_seqs; ++i) {
    for (int j = 0; j < i; ++j) {
      a[i * num_seqs + j] = rand() / (float)RAND_MAX;
      a[j * num_seqs + i] = a[i * num_seqs + j];
    }
    a[i * num_seqs + i] = INFINITY;
  }

  /*
  for (int i = 0; i < num_seqs; ++i) {
    for (int j = 0; j < num_seqs; ++j) {
      cout << a[num_seqs * i + j] << ",\t";
    }
    cout << "\n";
  }
  cout << "--------------------------------------\n";*/

  assert(num_seqs > 2);
  NJ nj((float *)a, num_seqs);
  nj.print();
  return 0;
}
